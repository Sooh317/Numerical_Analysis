#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int ITER = 100;

const long S = 1 << 12;
const int BLOCK_NUM = 1024;
const int THREAD_NUM = 256;
const int widthp = 4; // 1 << widthp 個のブロックが正方形の1行に対応

double elapsed(){
	struct timespec ts;
	clock_gettime(CLOCK_REALTIME, &ts);
	return ts.tv_sec + ts.tv_nsec*1.0e-9;
}

__global__
void kernel(int i, long n, long NN, int wp, int height, float r, float rr, float* a){
    float *dp = a + (i & 1)*NN, *ndp = a + (1 - (i & 1))*NN;
	long base = n + 1;
	long idx = base + (blockIdx.x >> wp) * n + (blockIdx.x & ((1 << wp) - 1)) * blockDim.x + threadIdx.x;
	while(idx < NN - n){
		ndp[idx] = rr*dp[idx] + r*(dp[idx - n] + dp[idx + n] + dp[idx - 1] + dp[idx + 1]);
	    idx += n * height;
	}
}


int main(){
    const long N = S + 2;
	const long NN = N*N;
	const int height = BLOCK_NUM / (S / THREAD_NUM);
	int size = (2*N*N) * sizeof(float);
	float* dp_host = (float*)malloc(size);
    const float r = 0.1f;
	const float rr = 1.0 - 4*r;

	for(int i = 0; i < N*N; i++){
        if(i < N || i >= N*(N-1) || i % N == 0 || i % N == N - 1) continue;
        dp_host[i] = 1.0;
    }
		
	float *dp_dev;
	hipMalloc((void**)&dp_dev,size);

	double t0 = elapsed();
	hipMemcpy(dp_dev,dp_host,size,hipMemcpyHostToDevice);

	double t1 = elapsed();
	
	for(int i = 0; i < ITER; i++){
		kernel<<<BLOCK_NUM,THREAD_NUM>>>(i, N, NN, widthp, height, r, rr, dp_dev);
	}
	hipDeviceSynchronize();

	double t2 = elapsed();

	hipMemcpy(dp_host,dp_dev,size,hipMemcpyDeviceToHost);

	double t3 = elapsed();

	for(int i = 1; i <= 10; i++){
	    for(int j = 1; j <= 10; j++){
            printf("%f ", dp_host[(ITER & 1)*(N*N) + i*N + j]);
        }
        printf("\n");
    }

	printf("H2D : %f\nCOMP: %f\nD2H : %f\n",t1-t0,t2-t1,t3-t2);

	hipFree(dp_dev);
	free(dp_host);
	return 0;
}
