#include <time.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

const int ITER = 100;
const int S = 10;

double elapsed(){
	struct timespec ts;
	clock_gettime(CLOCK_REALTIME, &ts);
	return ts.tv_sec + ts.tv_nsec*1.0e-9;
}

__global__
void kernel(int n, float r, float* a){
    int NN = n*n;
    float rr = 1 - 4*r;
    for(int i = 0; i < ITER; i++){
        float *dp = a + (i & 1)*NN, *ndp = a + (1 - (i & 1))*NN;
	    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	    while(idx < NN){
            bool dame = (idx - n < 0 || idx % n == 0 || idx % n == n - 1 || idx + n >= NN);
            if(!dame) ndp[idx] = rr*dp[idx] + r*(dp[idx - n] + dp[idx + n] + dp[idx - 1] + dp[idx + 1]);
		    idx += blockDim.x * gridDim.x;
	    }
        __syncthreads();
    }
}

/*
__global__
void kernel(int N, float r, float rr, float *u){
    int i = (threadIdx.x + 1) * 12;
    for(int t=1; t<=100; t++){
        int k = (t % 2) * N * N;
        int kk = (1 - (t % 2)) * N * N;
        for(int j=1; j<N-1; j++) u[k + i + j] = rr * u[kk + i + j] + r * (u[kk + i - N + j] + u[kk + i + j - 1] + u[kk + i + N + j] + u[kk + i + j + 1]);
        __syncthreads();
    } 
}
*/


int main(){
    const int N = S + 2;
	int size = (2*N*N) * sizeof(float);
	float* dp_host = (float*)malloc(size);
    const float r = 0.1f;

	for(int i = 0; i < N*N; i++){
        if(i < N || i >= N*(N-1) || i % N == 0 || i % N == N - 1) continue;
        dp_host[i] = 1.0;
    }
		
	float *dp_dev;
	hipMalloc((void**)&dp_dev,size);

	double t0 = elapsed();
	hipMemcpy(dp_dev,dp_host,size,hipMemcpyHostToDevice);

	double t1 = elapsed();
	
	kernel<<<1,16>>>(N, r, dp_dev);
	hipDeviceSynchronize();

	double t2 = elapsed();

	hipMemcpy(dp_host,dp_dev,size,hipMemcpyDeviceToHost);

	double t3 = elapsed();
	for(int i = 0; i < N; i++){
	    for(int j = 0; j < N; j++){
            printf("%f ", dp_host[(ITER & 1)*(N*N) + i*N + j]);
        }
        printf("\n");
    }

	printf("H2D : %f\nCOMP: %f\nD2H : %f\n",t1-t0,t2-t1,t3-t2);

	hipFree(dp_dev);
	free(dp_host);
	return 0;
}
